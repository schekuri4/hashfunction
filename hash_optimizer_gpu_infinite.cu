#include <iostream>
#include <vector>
#include <fstream>
#include <string>
#include <sstream>
#include <cmath>
#include <chrono>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iomanip>
#include <thread>
#include <atomic>
#include <signal.h>

using namespace std;

#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            cout << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - " << hipGetErrorString(error) << endl; \
            exit(1); \
        } \
    } while(0)

// Global variables for graceful shutdown
atomic<bool> keep_running(true);
double global_best_score = 1000.0;
unsigned int global_best_h = 0;
unsigned int global_best_k = 0;
string output_filename;
chrono::system_clock::time_point start_timestamp;

// Signal handler for Ctrl+C
void signal_handler(int signal) {
    cout << "\n\nReceived interrupt signal. Saving results and shutting down gracefully..." << endl;
    keep_running = false;
}

// New optimized GPU hash function
__device__ unsigned int hash_function_gpu(const char* text, int len, unsigned int h_seed, unsigned int k_seed) {
    unsigned int h = h_seed ^ 0x9e3779b9;
    unsigned int k = k_seed ^ 0x85ebca6b;
    
    for (int i = 0; i < len; i++) {
        unsigned int c = (unsigned int)text[i];
        
        h ^= c * 0x9e3779b1;
        k += c * 0xc2b2ae35;
        
        h = (h << 13) | (h >> 19);
        k = (k << 17) | (k >> 15);
        
        h += k * 0x165667b1;
        k ^= h * 0x27d4eb2f;
        
        h ^= k;
        k += h;
    }
    
    h ^= len;
    k ^= h;
    
    h ^= h >> 16;
    h *= 0x85ebca6b;
    h ^= h >> 13;
    h *= 0xc2b2ae35;
    h ^= h >> 16;
    
    return h;
}

// GPU kernel for calculating standard deviation for multiple datasets
__global__ void calculate_multi_dataset_std_dev_kernel(
    char** datasets, int** string_lengths, int** string_offsets, int* num_strings_per_dataset,
    int num_datasets, unsigned int* h_seeds, unsigned int* k_seeds, 
    double* results, int num_tests, int k) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_tests) return;
    
    unsigned int h_seed = h_seeds[idx];
    unsigned int k_seed = k_seeds[idx];
    
    double total_std_dev = 0.0;
    
    // Test on all datasets
    for (int dataset_idx = 0; dataset_idx < num_datasets; dataset_idx++) {
        // Use fixed-size array with safe maximum for GPU stack memory
        const int MAX_BUCKETS = 5000;  // Reduced to prevent stack overflow
        int buckets[MAX_BUCKETS];
        
        // Only use up to k buckets (bounded by MAX_BUCKETS)
        int actual_k = min(k, MAX_BUCKETS);
        for (int i = 0; i < actual_k; i++) buckets[i] = 0;
        
        int num_strings = num_strings_per_dataset[dataset_idx];
        
        // Hash all strings in this dataset
        for (int i = 0; i < num_strings; i++) {
            char* str_start = datasets[dataset_idx] + string_offsets[dataset_idx][i];
            int str_len = string_lengths[dataset_idx][i];
            unsigned int hash_val = hash_function_gpu(str_start, str_len, h_seed, k_seed);
            // Use unsigned arithmetic to avoid negative modulo issues
            unsigned int bucket = hash_val % actual_k;
            buckets[bucket]++;
        }
        
        // Calculate standard deviation for this dataset
        double mean = static_cast<double>(num_strings) / actual_k;
        double variance = 0.0;
        
        for (int i = 0; i < actual_k; i++) {
            double diff = buckets[i] - mean;
            variance += diff * diff;
        }
        variance /= actual_k;
        total_std_dev += sqrt(variance);
    }
    
    // Store average standard deviation across all datasets
    results[idx] = total_std_dev / num_datasets;
}

// GPU kernel for random seed generation
__global__ void generate_seeds_kernel(unsigned int* h_seeds, unsigned int* k_seeds, 
                                     int num_tests, unsigned long long seed_offset) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_tests) return;
    
    hiprandState state;
    hiprand_init(seed_offset + idx, 0, 0, &state);
    
    h_seeds[idx] = hiprand(&state);
    k_seeds[idx] = hiprand(&state);
}

// Structure for GPU dataset
struct GPUDataset {
    char* d_data;
    int* d_lengths;
    int* d_offsets;
    int num_strings;
    int total_chars;
};

GPUDataset prepare_dataset_for_gpu(const vector<string>& dataset) {
    GPUDataset gpu_dataset;
    gpu_dataset.num_strings = dataset.size();
    
    // Calculate total characters needed
    gpu_dataset.total_chars = 0;
    for (const string& str : dataset) {
        gpu_dataset.total_chars += str.length();
    }
    
    // Prepare host data
    vector<char> all_chars;
    vector<int> lengths;
    vector<int> offsets;
    
    int current_offset = 0;
    for (const string& str : dataset) {
        offsets.push_back(current_offset);
        lengths.push_back(str.length());
        
        for (char c : str) {
            all_chars.push_back(c);
        }
        current_offset += str.length();
    }
    
    // Allocate GPU memory
    CUDA_CHECK(hipMalloc(&gpu_dataset.d_data, gpu_dataset.total_chars * sizeof(char)));
    CUDA_CHECK(hipMalloc(&gpu_dataset.d_lengths, gpu_dataset.num_strings * sizeof(int)));
    CUDA_CHECK(hipMalloc(&gpu_dataset.d_offsets, gpu_dataset.num_strings * sizeof(int)));
    
    // Copy to GPU
    CUDA_CHECK(hipMemcpy(gpu_dataset.d_data, all_chars.data(), 
                         gpu_dataset.total_chars * sizeof(char), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(gpu_dataset.d_lengths, lengths.data(), 
                         gpu_dataset.num_strings * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(gpu_dataset.d_offsets, offsets.data(), 
                         gpu_dataset.num_strings * sizeof(int), hipMemcpyHostToDevice));
    
    return gpu_dataset;
}

void free_gpu_dataset(GPUDataset& dataset) {
    hipFree(dataset.d_data);
    hipFree(dataset.d_lengths);
    hipFree(dataset.d_offsets);
}

vector<string> load_dataset(const string& filename) {
    vector<string> dataset;
    ifstream file(filename);
    string line;
    
    while (getline(file, line)) {
        if (!line.empty()) {
            dataset.push_back(line);
        }
    }
    
    return dataset;
}

string extract_hash_function() {
    ifstream source_file(__FILE__);
    string line;
    string hash_function = "";
    bool in_function = false;
    int brace_count = 0;
    
    while (getline(source_file, line)) {
        // Look for the start of the hash function
        if (line.find("__device__ unsigned int hash_function_gpu") != string::npos) {
            in_function = true;
            hash_function += line + "\n";
            if (line.find("{") != string::npos) {
                brace_count = 1;
            }
            continue;
        }
        
        if (in_function) {
            hash_function += line + "\n";
            
            // Count braces to find the end of the function
            for (char c : line) {
                if (c == '{') brace_count++;
                else if (c == '}') brace_count--;
            }
            
            // If we've closed all braces, we're done
            if (brace_count == 0) {
                break;
            }
        }
    }
    
    source_file.close();
    return hash_function;
}

void save_best_result(double best_score, unsigned int best_h, unsigned int best_k, 
                     long long total_tests, int runtime_seconds) {
    // Get current time for this result
    auto now = chrono::system_clock::now();
    auto time_t = chrono::system_clock::to_time_t(now);
    
    // Append to the single output file
    ofstream file(output_filename, ios::app);
    file << "[" << put_time(localtime(&time_t), "%Y-%m-%d %H:%M:%S") << "] "
         << "GPU NEW BEST: Score=" << fixed << setprecision(6) << best_score 
         << " | Hash: h=" << best_h << "(0x" << hex << best_h << dec 
         << "), k=" << best_k << "(0x" << hex << best_k << dec 
         << ") | Tests=" << total_tests << " | Runtime=" << runtime_seconds 
         << "s | Rate=" << (total_tests / max(1, runtime_seconds)) << "/s" << endl;
    file.close();
}

int main() {
    // Set up signal handler for graceful shutdown
    signal(SIGINT, signal_handler);
    
    cout << "=== INFINITE GPU HASH FUNCTION OPTIMIZER ===" << endl;
    cout << "Press Ctrl+C to stop and save the best result found" << endl;
    
    // Check CUDA availability
    int device_count;
    CUDA_CHECK(hipGetDeviceCount(&device_count));
    if (device_count == 0) {
        cout << "No CUDA devices found!" << endl;
        return 1;
    }
    
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    cout << "\nUsing GPU: " << prop.name << endl;
    cout << "Compute capability: " << prop.major << "." << prop.minor << endl;
    cout << "Max threads per block: " << prop.maxThreadsPerBlock << endl;
    cout << "Global memory: " << (prop.totalGlobalMem / 1024 / 1024) << " MB" << endl;
    
    // Load datasets - include ALL input files
    vector<string> dataset_names = {
        "inputs/1.txt",
        "inputs/2.txt",
        "inputs/3.txt",
        "inputs/4.txt",
        "inputs/5.txt",
        "inputs/atoz.txt",
        "inputs/bertuncased.txt",
        "inputs/common500.txt",
        "inputs/mit_a.txt",
        "inputs/sample_input.txt",
        "inputs/test_alls.txt",
        "inputs/test_passwords.txt",
        "inputs/test_wordle500.txt"
    };
    
    vector<vector<string>> datasets;
    vector<GPUDataset> gpu_datasets;
    
    cout << "\nLoading datasets..." << endl;
    for (const string& name : dataset_names) {
        vector<string> dataset = load_dataset(name);
        if (dataset.empty()) {
            cout << "Warning: Could not load " << name << endl;
            continue;
        }
        datasets.push_back(dataset);
        gpu_datasets.push_back(prepare_dataset_for_gpu(dataset));
        cout << "Loaded " << dataset.size() << " entries from " << name << endl;
    }
    
    // Initialize output file with start timestamp
    start_timestamp = chrono::system_clock::now();
    auto start_time_t = chrono::system_clock::to_time_t(start_timestamp);
    
    stringstream filename;
    filename << "hash_optimization_results_" 
             << put_time(localtime(&start_time_t), "%Y%m%d_%H%M%S") << ".txt";
    output_filename = filename.str();
    
    ofstream file(output_filename);
    file << "=== HASH FUNCTION OPTIMIZATION RESULTS ===" << endl;
    file << "Started: " << put_time(localtime(&start_time_t), "%Y-%m-%d %H:%M:%S") << endl;
    file << "Hash Function Implementation:" << endl;
    file << extract_hash_function();
    file << "Datasets: " << datasets.size() << " loaded" << endl;
    file << "=== OPTIMIZATION PROGRESS ===" << endl;
    file.close();
    
    cout << "\nResults will be saved to: " << output_filename << endl;
    
    if (datasets.empty()) {
        cout << "Error: No datasets loaded!" << endl;
        return 1;
    }
    
    // Calculate bucket count based on total number of strings
    int total_strings = 0;
    for (const auto& dataset : datasets) {
        total_strings += dataset.size();
    }
    int k = total_strings;  // Use total number of strings as bucket count
    
    // Ensure bucket count is within GPU memory limits
    if (k > 1000) {
        k = 1000;  // Cap at 1000 for GPU memory safety
        cout << "\nNote: Bucket count capped at 1000 for GPU memory safety" << endl;
    }
    
    cout << "\nAutomatic bucket calculation:" << endl;
    cout << "- Total strings across all datasets: " << total_strings << endl;
    cout << "- Using " << k << " buckets for optimization" << endl;
    
    // Configuration for maximum GPU utilization
    int tests_per_batch = 1000000;  // 1M tests per batch
    int threads_per_block = 256;
    int blocks = (tests_per_batch + threads_per_block - 1) / threads_per_block;
    
    cout << "\nGPU Configuration:" << endl;
    cout << "- Tests per batch: " << tests_per_batch << endl;
    cout << "- Threads per block: " << threads_per_block << endl;
    cout << "- Number of blocks: " << blocks << endl;
    cout << "- Datasets: " << datasets.size() << endl;
    
    // Prepare GPU memory for multi-dataset processing
    vector<char*> h_dataset_ptrs(gpu_datasets.size());
    vector<int*> h_lengths_ptrs(gpu_datasets.size());
    vector<int*> h_offsets_ptrs(gpu_datasets.size());
    vector<int> h_num_strings(gpu_datasets.size());
    
    for (size_t i = 0; i < gpu_datasets.size(); i++) {
        h_dataset_ptrs[i] = gpu_datasets[i].d_data;
        h_lengths_ptrs[i] = gpu_datasets[i].d_lengths;
        h_offsets_ptrs[i] = gpu_datasets[i].d_offsets;
        h_num_strings[i] = gpu_datasets[i].num_strings;
    }
    
    // Allocate GPU memory for dataset pointers
    char** d_dataset_ptrs;
    int** d_lengths_ptrs;
    int** d_offsets_ptrs;
    int* d_num_strings;
    
    CUDA_CHECK(hipMalloc(&d_dataset_ptrs, gpu_datasets.size() * sizeof(char*)));
    CUDA_CHECK(hipMalloc(&d_lengths_ptrs, gpu_datasets.size() * sizeof(int*)));
    CUDA_CHECK(hipMalloc(&d_offsets_ptrs, gpu_datasets.size() * sizeof(int*)));
    CUDA_CHECK(hipMalloc(&d_num_strings, gpu_datasets.size() * sizeof(int)));
    
    CUDA_CHECK(hipMemcpy(d_dataset_ptrs, h_dataset_ptrs.data(), 
                         gpu_datasets.size() * sizeof(char*), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_lengths_ptrs, h_lengths_ptrs.data(), 
                         gpu_datasets.size() * sizeof(int*), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_offsets_ptrs, h_offsets_ptrs.data(), 
                         gpu_datasets.size() * sizeof(int*), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_num_strings, h_num_strings.data(), 
                         gpu_datasets.size() * sizeof(int), hipMemcpyHostToDevice));
    
    // Allocate GPU memory for seeds and results
    unsigned int *d_h_seeds, *d_k_seeds;
    double *d_results;
    
    CUDA_CHECK(hipMalloc(&d_h_seeds, tests_per_batch * sizeof(unsigned int)));
    CUDA_CHECK(hipMalloc(&d_k_seeds, tests_per_batch * sizeof(unsigned int)));
    CUDA_CHECK(hipMalloc(&d_results, tests_per_batch * sizeof(double)));
    
    // Host memory for results
    vector<double> h_results(tests_per_batch);
    vector<unsigned int> h_h_seeds(tests_per_batch);
    vector<unsigned int> h_k_seeds(tests_per_batch);
    
    auto start_time = chrono::high_resolution_clock::now();
    long long total_tests = 0;
    int batch_count = 0;
    unsigned long long seed_offset = 0;
    
    cout << "\n🚀 Starting infinite optimization..." << endl;
    cout << "Current best: " << fixed << setprecision(6) << global_best_score << endl;
    
    while (keep_running) {
        batch_count++;
        
        // Generate random seeds
        generate_seeds_kernel<<<blocks, threads_per_block>>>(
            d_h_seeds, d_k_seeds, tests_per_batch, seed_offset);
        CUDA_CHECK(hipDeviceSynchronize());
        
        // Run optimization kernel
        calculate_multi_dataset_std_dev_kernel<<<blocks, threads_per_block>>>(
            d_dataset_ptrs, d_lengths_ptrs, d_offsets_ptrs, d_num_strings,
            gpu_datasets.size(), d_h_seeds, d_k_seeds, d_results, tests_per_batch, k);
        CUDA_CHECK(hipDeviceSynchronize());
        
        // Copy results back to host
        CUDA_CHECK(hipMemcpy(h_results.data(), d_results, 
                             tests_per_batch * sizeof(double), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(h_h_seeds.data(), d_h_seeds, 
                             tests_per_batch * sizeof(unsigned int), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(h_k_seeds.data(), d_k_seeds, 
                             tests_per_batch * sizeof(unsigned int), hipMemcpyDeviceToHost));
        
        // Find best result in this batch
        auto min_it = min_element(h_results.begin(), h_results.end());
        int best_idx = distance(h_results.begin(), min_it);
        double batch_best = *min_it;
        
        total_tests += tests_per_batch;
        
        // Update global best if we found something better
        if (batch_best < global_best_score) {
            global_best_score = batch_best;
            global_best_h = h_h_seeds[best_idx];
            global_best_k = h_k_seeds[best_idx];
            
            auto current_time = chrono::high_resolution_clock::now();
            auto duration = chrono::duration_cast<chrono::seconds>(current_time - start_time);
            
            // Get current timestamp
            auto now = chrono::system_clock::now();
            auto time_t = chrono::system_clock::to_time_t(now);
            
            cout << "\n🎉 [" << put_time(localtime(&time_t), "%Y-%m-%d %H:%M:%S") 
                 << "] NEW BEST: Score=" << fixed << setprecision(6) << global_best_score 
                 << " | Hash: h=" << global_best_h << "(0x" << hex << global_best_h << dec 
                 << "), k=" << global_best_k << "(0x" << hex << global_best_k << dec 
                 << ") | Function: h=h_seed^0x9e3779b9, k=k_seed^0x85ebca6b | Tests=" << total_tests 
                 << " | Runtime=" << duration.count() << "s | Rate=" 
                 << (total_tests / max(1, (int)duration.count())) << "/s" << endl;
            
            // Save immediately when we find a better result
            save_best_result(global_best_score, global_best_h, global_best_k, 
                           total_tests, duration.count());
        }
        
        // Progress update every 10 batches
        if (batch_count % 10 == 0) {
            auto current_time = chrono::high_resolution_clock::now();
            auto duration = chrono::duration_cast<chrono::seconds>(current_time - start_time);
            
            cout << "Batch " << batch_count << " | " << total_tests << " tests | " 
                 << duration.count() << "s | Best: " << fixed << setprecision(6) 
                 << global_best_score << " | Rate: " 
                 << (total_tests / max(1, (int)duration.count())) << " tests/sec" << endl;
        }
        
        seed_offset += tests_per_batch;
    }
    
    // Final save and cleanup
    auto end_time = chrono::high_resolution_clock::now();
    auto total_duration = chrono::duration_cast<chrono::seconds>(end_time - start_time);
    
    cout << "\n=== INFINITE OPTIMIZATION STOPPED ===" << endl;
    cout << "Total runtime: " << total_duration.count() << " seconds" << endl;
    cout << "Total tests: " << total_tests << endl;
    cout << "Average tests per second: " << (total_tests / max(1, (int)total_duration.count())) << endl;
    
    cout << "\n🏆 FINAL BEST RESULT:" << endl;
    cout << "h_seed: " << global_best_h << " (0x" << hex << global_best_h << dec << ")" << endl;
    cout << "k_seed: " << global_best_k << " (0x" << hex << global_best_k << dec << ")" << endl;
    cout << "Average Standard Deviation: " << fixed << setprecision(6) << global_best_score << endl;
    
    save_best_result(global_best_score, global_best_h, global_best_k, 
                    total_tests, total_duration.count());
    
    // Cleanup
    for (auto& gpu_dataset : gpu_datasets) {
        free_gpu_dataset(gpu_dataset);
    }
    
    hipFree(d_dataset_ptrs);
    hipFree(d_lengths_ptrs);
    hipFree(d_offsets_ptrs);
    hipFree(d_num_strings);
    hipFree(d_h_seeds);
    hipFree(d_k_seeds);
    hipFree(d_results);
    
    return 0;
}